/*
Name: John Stephenson
BlazerId: johnds39
Course Section: CS 432
Homework #: 5
*/


#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <iostream>

using namespace std;

const int ALIVE = 1;
const int DEAD = 0;

/* prints the table passed into the the function and its generation */
void printTable(int* table, int gen, int N2){
	cout << "Generation " << gen << ":\n";
	for (int i = 0; i < N2; i++) {
		for (int j = 0; j < N2; j++) {
			cout << table[N2 * i + j] << " ";
		}
		cout << "\n";
	}
}

/* modifies the nextGen table to represent the next generation of The Game of Life */
__global__
void nextGeneration(int* table, int* nextGen, int N2) {

	int j = blockDim.x * blockIdx.x + threadIdx.x + 1;
	
	for (int i = 1; i < N2-1; i++){
		if (j < N2-1) {
			int localChange = 0;
			int neighbors = 0;
			neighbors += table[N2 * (i-1) + j];
			neighbors += table[N2 * (i-1) + (j-1)];
			neighbors += table[N2 * (i-1) + (j+1)];
			neighbors += table[N2 * (i+1) + j];
			neighbors += table[N2 * (i+1) + (j-1)];
			neighbors += table[N2 * (i+1) + (j+1)];
			neighbors += table[N2 * i + (j+1)];
			neighbors += table[N2 * i + (j-1)];
	
			if (table[N2 * i + j] == DEAD && neighbors == 3) {
				nextGen[N2 * i + j] = ALIVE;
				localChange = 1;
			}
	
			if (neighbors <= 1 || neighbors >= 4) {
				if (table[N2 * i + j] == ALIVE) {
					nextGen[N2 * i + j] = DEAD;
					localChange = 1;
				}
			}

			/* this is used to make sure the two tables stay up to date with each other over the generations since they are being swapped after each iteration */
			if(localChange == 0) {
				nextGen[N2 * i + j] = table[N2 * i + j];
			}
	
		}
	}
}

/* initializes a table according to the size provided by the user with each element being randomized to be alive or dead */
void initTable(int* table, int N2){
	for (int i = 0; i < N2; i++) {
		for (int j = 0; j < N2; j++) {
			if (i == N2 - 1 || j == N2 - 1 || i == 0 || j == 0) {
				table[N2 * i + j] = DEAD;
			}
			else {
				if (rand() % 2 < 1) {
					table[N2 * i + j] = ALIVE;
				}
				else {
					table[N2 * i + j] = DEAD;
				}
			}

		}
	}
}

int main(int argc, char *argv[]){
	clock_t starttime, endtime;
	int N = atoi(argv[1]);
	int maxGen = atoi(argv[2]);
	srand(time(NULL));

	//freopen("output2.txt", "w", stdout);
	
	const int N2 = N + 2;

	int *table;
	int *nextGen;

	hipMallocManaged((void **)&table, N2 * N2 * sizeof(int));
	hipMallocManaged((void **)&nextGen, N2 * N2 * sizeof(int));

	initTable(table, N2);

	/* copying the initial values of the table into the nextGen table */
	for (int i = 0; i < N2; i++) {
		for (int j = 0; j < N2; j++) {
			nextGen[N2 * i + j] = table[N2 * i + j];
		}
	}

	int blockSize = 128;
	int numBlocks = (N + blockSize - 1) / blockSize;

	starttime = clock();

	/* the main game loop that continues until the max generation or the game over condition has been met */
	for(int i = 0; i < maxGen; i++){
		//printTable(table, i, N2);
		nextGeneration <<<numBlocks, blockSize>>> (table, nextGen, N2);
		hipDeviceSynchronize();
		swap(table, nextGen);
	}

	endtime = clock();
	printf("Time taken = %lf seconds\n", ((double) endtime - starttime) / CLOCKS_PER_SEC);

	hipFree(table);
	hipFree(nextGen);

	return 0;
}